#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#define M 512
#define K 512
#define N 512

#define BLOCK_SIZE 32 // block size, each thread to calcualte each block

void initial(float *array, int size){
    for(size_t i=0; i<size; i++){
        array[i] = (float)(rand()%10+1);
    }
    return;
}

void printMatrix(float *array, int row, int col){
    float *p=array;
    for(size_t y=0; y<row; y++){
        for(size_t x=0; x<col; x++){
            printf("%10lf",p[x]);
        }
        p = p + col;
        printf("\n");
    }
    return;
}

void multiplicateMatrixOnHost(float *array_A, 
                              float *array_B, 
                              float *array_C, 
                              int M_p, 
                              int K_p, 
                              int N_p){
    for(size_t i=0; i<M_p; i++){
        for(size_t j=0; j<N_p; j++){
            float sum = 0;
            for(size_t k=0; k<K_p; k++){
                sum += array_A[i*K_p+k]*array_B[k*N_p+j];
            }
            array_C[i*N_p+j] = sum;
        }
    }
    return;
}

__global__ void multiplicateMatrixOnDevice(float *array_A, 
                                           float *array_B, 
                                           float *array_C,
                                           int M_p,
                                           int K_p,
                                           int N_p){
    int ix = threadIdx.x+blockIdx.x*blockDim.x; // row index
    int iy = threadIdx.y+blockIdx.y*blockDim.y; // col index
    if(ix<N_p && iy<M_p){
        float sum=0;
        for(size_t k=0; k<K_p; k++){
            sum += array_A[iy*K_p+k]*array_B[k*N_p+ix];
        }
        array_C[iy*M_p+ix] = sum;
    }
    return;
}

__global__ void matrixMultiplyShared(float *A, 
                                     float *B, 
                                     float *C, 
                                     int numARows,
                                     int numAColumns,
                                     int numBRows,
                                     int numBColumns,
                                     int numCRows,
                                     int numCColumns){

    __shared__ float sharedM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedN[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockidx.x;
    int by = blockidy.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by*BLOCK_SIZE+ty;
    int col = bx*BLOCK_SIZE+tx;

    float Csub = 0.0;

    for(size_t i=0; i<(int)(ceil((float)numAColumns)/BLOCK_SIZE); i++){
        if(i*BLOCK_SIZE + tx < numAColumns && row < numARows)
            sharedM[ty][tx] = A[row*numAColumns + i*BLOCK_SIZE + tx];
        else
            sharedM[ty][tx] = 0.0;
        if(i*BLOCK_SIZE + ty <numBRows && col < numBColumns)
            sharedN[ty][tx] = B[(i*BLOCK_SIZE + ty)*numBColumns + col];
        else
            sharedN[ty][tx] = 0.0;
        
        __syncthreads();

        for(int j=0; j<BLOCK_SIZE; j++)
            Csub += shareM[ty][j]*sharedN[j][tx];
        
        __syncthreads();
    }
    if(row < numCRows && col < numCColumns)
        C[row*numCColumns + col] = Csub;
}


int main(int argc, char **argv){
    clock_t start = 0, finish = 0;
    float time;

    int Axy = M*K;
    int Bxy = K*N;
    int Cxy = M*N;

    float *h_A, *h_B, *hostRef, *deviceRef;
    h_A = (float*)malloc(Axy*sizeof(float));
    h_B = (float*)malloc(Bxy*sizeof(float));

    int nBytes = M*N*sizeof(float);
    hostRef = (float*)malloc(Cxy*sizeof(float));
    deviceRef = (float*)malloc(Cxy*sizeof(float));

    initial(h_A, Axy);
    initial(h_B, Bxy);

    start = clock();
    multiplicateMatrixOnHost(h_A, h_B, hostRef, M, K, N);
    finish = clock();
    printf("\n");
    printf("------------------------------------------------------\n");
    printf("Computing matrix product using multiplicateMatrixOnHost \n");
    printf("------------------------------------------------------\n");
    printf("Matrix_hostRef:(%dx%d) CPU run time is :%lfs\n", M, N, time);

    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, Axy*sizeof(float));
    hipMalloc((void**)&d_B, Bxy*sizeof(float));
    hipMalloc((void**)&d_C, Cxy*sizeof(float));

    hipMemcpy(d_A, h_A, Axy*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, Axy*sizeof(float), hipMemcpyHostToDevice);


}
