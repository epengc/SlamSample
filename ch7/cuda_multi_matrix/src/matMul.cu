#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#define M 512
#define K 512
#define N 512
// Tread block size
#define BLOCK_SIZE 16

typedef struct{
    int width;
    int height;
    int stride;
    float *elements;
}Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col){
    return A.elements[row*A.stride+col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value){
    A.elements[row*A.stride+col]=value;
}

// Get the BLOCK_SIZE \times BLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down 
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col){
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[row*A.stride*BLOCK_SIZE+BLOCK_SIZE*col];
    return Asub;
}


// Forward declaration - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

void MatMul(const Matrix A, const Matrix B, Matrix C){
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.stride = A.width;
    d_A.height = A.height;
    size_t size = A.width*A.height*sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width;
    d_B.stride = B.width;
    d_B.height = B.height;
    size = B.width*B.height*sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    //Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.stide = C.width;
    d_C.height = C.height;
    size = C.width*C.height*sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from Device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device Memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
    // Block row and Column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    for(int m=0; m<(A.width/BLOCK_SIZE); ++m){
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        __share__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __share__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        __syncthreads();

        for(int e=0; e<BLOCK_SIZE; ++e)
            Cvalue += As[row][e]*Bs[e][col];
        
        __syncthreads();
        }
    SetElement(Csub, row, col, Cvalue);
}
